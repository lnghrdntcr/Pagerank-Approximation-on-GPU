#include "hip/hip_runtime.h"
//
// Created by Francesco Sgherzi on 15/04/19.
//

#include <iostream>
#include <map>
#include <stdlib.h>
#include <time.h>
#include <vector>
#include <fstream>
#include <set>
#include <math.h>
#include <algorithm>
#include "Parse/Parse.h"

#define TAU 0.0
#define ALPHA 0.85

#define MAX_B 1024
#define MAX_T 1024

#define MAX_ITER 200

#define num_type double

using namespace std;

template<typename T>
void generate_sparse_matrix(T *matrix, const unsigned int DIMV, const unsigned int min_sparse) {

    // for all rows
    for (int i = 0; i < DIMV; ++i) {

        int num_zeroes = rand() % (DIMV - min_sparse) + min_sparse;
        std::set<int> zero_idxs;

        zero_idxs.insert(i);
        for (int j = 0; j < num_zeroes; ++j) {
            int r_idx = rand() % DIMV;
            zero_idxs.insert(r_idx);
        }

        for (int j = 0; j < DIMV; ++j) {
            if (zero_idxs.find(j) == zero_idxs.end() && (DIMV - zero_idxs.size()) != 0) {
                matrix[i * DIMV + j] = (T) 1.0 / (DIMV - zero_idxs.size());
            }
        }

    }
}

template<typename T>
void fill_spm(T *matrix, const unsigned int DIMV) {
    for (int i = 0; i < DIMV; ++i) {
        int count_zero = 0;
        for (int j = 0; j < DIMV; ++j) {
            if (matrix[i * DIMV + j] == 0.0) count_zero++;
        }
        if (count_zero == DIMV) matrix[i * DIMV + i] = 1;
    }
}

template<typename T>
void transpose(T *out, T *in, const unsigned DIMV) {

    for (int i = 0; i < DIMV; ++i) {
        for (int j = 0; j < DIMV; ++j) {
            out[i * DIMV + j] = in[j * DIMV + i];
        }
    }

}

template<typename T>
void to_csc(T *csc_val, int *csc_non_zero, int *csc_col_idx, T *src, const unsigned DIMV, const unsigned non_zero) {

    unsigned val_idx = 0;

    csc_non_zero[0] = 0;

    for (int i = 0; i < DIMV; ++i) {

        csc_non_zero[i + 1] = csc_non_zero[i];

        for (int j = 0; j < DIMV; ++j) {

            if (src[i * DIMV + j] > 0) {
                csc_val[val_idx] = src[i * DIMV + j];
                csc_non_zero[i + 1]++;
                csc_col_idx[val_idx] = j;

                val_idx++;
            }

        }

    }

    cout << "Bella" << endl;

}

template<typename T>
void to_device_csc(T *csc_val, int *csc_non_zero, int *csc_col_idx, const csc_t src) {

    hipMemcpy(csc_val, &src.val[0], sizeof(T) * src.val.size(), hipMemcpyHostToDevice);
    hipMemcpy(csc_non_zero, &src.non_zero[0], sizeof(int) * src.non_zero.size(), hipMemcpyHostToDevice);
    hipMemcpy(csc_col_idx, &src.col_idx[0], sizeof(int) * src.col_idx.size(), hipMemcpyHostToDevice);

}

template<typename T>
unsigned int count_non_zero(T *m, const unsigned int DIMV) {
    int sum = 0;

    for (int i = 0; i < DIMV * DIMV; ++i) {
        if (m[i] > 0) sum++;
    }

    return sum;
}

template<typename T>
__global__
void d_set_val(T *m, T value, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {

        for (int i = init; i < DIMV; i += stride) {
            m[i] = value;
        }

    }

}

template<typename T>
bool check_error(T *e, const T error, const unsigned DIMV) {
    for (int i = 0; i < DIMV; ++i) {
        if (e[i] > error) return false;
    }
    return true;
}

template<typename T>
__global__
void spmv(T *Y, T *pr, T *csc_val, int *csc_non_zero, int *csc_col_idx, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {

            int begin = csc_non_zero[i];
            int end = csc_non_zero[i + 1];

            T acc = 0.0;

            for (int j = begin; j < end; j++) {
                acc += csc_val[j] * pr[csc_col_idx[j]];
            }

            Y[i] = acc;

        }
    }

}

template<typename T>
__global__
void scale(T *m, T v, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {
            m[i] *= v;
        }
    }
}

template<typename T>
__global__
void shift(T *m, T v, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {
            m[i] += v;
        }
    }
}

template<typename T>
__global__
void compute_error(T *error, T *next, T *prev, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {
            error[i] = abs(next[i] - prev[i]);
        }
    }

}

__global__
void d_set_dangling_bitmap(bool *dangling_bitmap, int *csc_col_idx, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV; i += stride) {
        dangling_bitmap[csc_col_idx[i]] = 0;
    }

}


/**
 * Cannot use thrust's implementation of dot product because it goes out of memory
 * even for 100k pages.
 */
/*
T2 dot(size_t n, T1 *x, T2 *y){
    T2 result = thrust::inner_product(
            thrust::device_pointer_cast(x),
            thrust::device_pointer_cast(x + n),
            thrust::device_pointer_cast(y),
            0.0f);
    return result;
}
*/

template<typename T1, typename T2>
T2 dot(size_t n, T1 *x, T2 *y) {

    T1 *tempx;
    T2 *tempy;
    T2 result = 0.0;

    hipHostMalloc(&tempx, n * sizeof(T1));
    hipHostMalloc(&tempy, n * sizeof(T2));

    hipMemcpy(tempx, x, n * sizeof(T1), hipMemcpyDeviceToHost);
    hipMemcpy(tempy, y, n * sizeof(T2), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; ++i) {

        result += tempx[i] * tempy[i];

    }

    return result;

}

int main() {


    /**
     * HOST
     */
    num_type *matrix;
    num_type *matrix_t;
    num_type *pr;
    num_type *spmv_res;
    num_type *error;
    num_type *csc_val;
    int *csc_non_zero;
    int *csc_col_idx;


    /**
     * DEVICE
     */
    num_type *d_pr;
    num_type *d_error;
    num_type *d_spmv_res;
    num_type *d_csc_val;
    int *d_csc_non_zero;
    int *d_csc_col_idx;
    bool *d_dangling_bitmap;

    csc_t csc_matrix = parse_dir("/home/fra/University/HPPS/Approximate-PR/graph_generator/generated_csc/test");

    const unsigned NON_ZERO = csc_matrix.val.size();
    const unsigned DIM = csc_matrix.non_zero.size() - 1;

    std::cout << "\nFEATURES: " << std::endl;
    std::cout << "\tNumber of non zero elements: " << NON_ZERO << std::endl;
    std::cout << "\tNumber of nodes: " << DIM << std::endl;
    std::cout << "\tSparseness: " << (1 - (((double) NON_ZERO) / (DIM * DIM))) * 100 << "%\n" << std::endl;

    hipHostMalloc(&matrix, sizeof(num_type) * DIM * DIM);
    hipHostMalloc(&matrix_t, sizeof(num_type) * DIM * DIM);
    hipHostMalloc(&pr, sizeof(num_type) * DIM);
    hipHostMalloc(&spmv_res, sizeof(num_type) * DIM);
    hipHostMalloc(&error, sizeof(num_type) * DIM);


    std::cout << "Initializing device memory" << std::endl;

    // Create device memory
    hipMalloc(&d_csc_val, sizeof(num_type) * NON_ZERO);
    hipMalloc(&d_csc_non_zero, sizeof(int) * (DIM + 1));
    hipMalloc(&d_csc_col_idx, sizeof(num_type) * NON_ZERO);
    hipMalloc(&d_pr, sizeof(num_type) * DIM);
    hipMalloc(&d_error, sizeof(num_type) * DIM);
    hipMalloc(&d_spmv_res, sizeof(num_type) * DIM);
    hipMalloc(&d_dangling_bitmap, DIM * sizeof(bool));

    std::cout << "Parsing csc files" << std::endl;

    to_device_csc(d_csc_val, d_csc_non_zero, d_csc_col_idx, csc_matrix);

    std::cout << "Initializing pr, error, dangling bitmap vectors" << std::endl;

    // Initialize error and pr vector
    d_set_val << < MAX_B, MAX_T >> > (d_pr, 1.0 / DIM, DIM);
    d_set_val << < MAX_B, MAX_T >> > (d_error, 1.0, DIM);
    d_set_val << < MAX_B, MAX_T >> > (d_dangling_bitmap, true, DIM);

    d_set_dangling_bitmap << < MAX_B, MAX_T >> > (d_dangling_bitmap, d_csc_col_idx, NON_ZERO);

    //d_set_dangling_bitmap(d_dangling_bitmap, d_csc_col_idx, NON_ZERO);


    // Copy them back to their host vectors
    hipMemcpy(pr, d_pr, DIM * sizeof(num_type), hipMemcpyDeviceToHost);
    hipMemcpy(error, d_error, DIM * sizeof(num_type), hipMemcpyDeviceToHost);


    /**
    * TEST
    */

    /*   bool *dbm;

       hipHostMalloc(&dbm, DIM * sizeof(bool));
       hipMemcpy(dbm, d_dangling_bitmap, DIM * sizeof(bool), hipMemcpyDeviceToHost);

       for (int j = 0; j < DIM; ++j) {
           std::cout << dbm[j] << std::endl;
       }*/


    /**
     * END TEST
     */

    std::cout << "Beginning pagerank..." << std::endl;

    int iterations = 0;
    while (!check_error(error, TAU, DIM) && iterations < MAX_ITER) {

        // TODO: andare a guardare quali sono i valori ottimali sulla gpu
        spmv << < MAX_B, MAX_T >> > (d_spmv_res, d_pr, d_csc_val, d_csc_non_zero, d_csc_col_idx, DIM);
        scale << < MAX_B, MAX_T >> > (d_spmv_res, ALPHA, DIM);

        hipDeviceSynchronize();

        num_type res_v = dot(DIM, d_dangling_bitmap, d_pr);

        shift << < MAX_B, MAX_T >> > (d_spmv_res, (1.0 - ALPHA) / DIM + (ALPHA / DIM) * res_v, DIM);

        compute_error << < MAX_B, MAX_T >> > (d_error, d_spmv_res, d_pr, DIM);

        hipDeviceSynchronize();

        hipMemcpy(error, d_error, DIM * sizeof(num_type), hipMemcpyDeviceToHost);
        hipMemcpy(d_pr, d_spmv_res, DIM * sizeof(num_type), hipMemcpyDeviceToDevice);

        iterations++;
    }

    hipMemcpy(pr, d_pr, DIM * sizeof(num_type), hipMemcpyDeviceToHost);


    std::cout << "converged after n_iter: " << iterations << std::endl;


    std::map<int, num_type> pr_map;
    std::vector<std::pair<int, num_type>> sorted_pr;
    std::vector<int> sorted_pr_idxs;

    for (int i = 0; i < DIM; ++i) {
        sorted_pr.push_back({i, pr[i]});
        pr_map[i] = pr[i];
        //std::cout << "Index: " << i << " => " << pr_map[i] << std::endl;
    }

    std::sort(sorted_pr.begin(), sorted_pr.end(),
              [](const std::pair<int, num_type> &l, const std::pair<int, num_type> &r) {
                  return l.second > r.second;
              });

    // print the vector
    for (auto const &pair: sorted_pr) {
        sorted_pr_idxs.push_back(pair.first);
    }

    std::cout << "Checking results..." << std::endl;

    std::ifstream results;
    results.open("/home/fra/University/HPPS/Approximate-PR/graph_generator/generated_csc/test/results.txt");

    int i = 0;
    int tmp = 0;

    while (results >> tmp) {
        if (tmp != sorted_pr_idxs[i]) {
            std::cout << "ERROR AT INDEX " << i << ": " << tmp << " != " << sorted_pr_idxs[i] << " Value => "
                      << (double) pr_map[sorted_pr_idxs[i]] << std::endl;
        }
        i++;
    }


    hipFree(&matrix);
    hipFree(&matrix_t);
    hipFree(&pr);
    hipFree(&spmv_res);
    hipFree(&error);
    hipFree(&csc_val);
    hipFree(&csc_non_zero);
    hipFree(&csc_col_idx);

    hipFree(&d_pr);
    hipFree(&d_error);
    hipFree(&d_spmv_res);
    hipFree(&d_csc_val);
    hipFree(&d_csc_non_zero);
    hipFree(&d_csc_col_idx);

    return 0;
}
