#include "hip/hip_runtime.h"
// Created by Francesco Sgherzi on 15/04/19.
//

#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <iostream>
#include <fstream>
#include <map>
#include <vector>
#include <algorithm>

#include <thrust/inner_product.h>

#include "Parse/Parse.h"
#include "Utils/Utils.h"

#define TAU 0.0
#define ALPHA 0.85

#define MAX_B 1024
#define MAX_T 1024

#define MAX_ITER 200

#define num_type double

template<typename T>
bool check_error(T *e, const T error, const unsigned DIMV) {
    for (int i = 0; i < DIMV; ++i) {
        if (e[i] > error) return false;
    }
    return true;
}

template<typename T>
void to_device_csc(T *csc_val, int *csc_non_zero, int *csc_col_idx, const csc_t src) {

    hipMemcpy(csc_val, &src.val[0], sizeof(T) * src.val.size(), hipMemcpyHostToDevice);
    hipMemcpy(csc_non_zero, &src.non_zero[0], sizeof(int) * src.non_zero.size(), hipMemcpyHostToDevice);
    hipMemcpy(csc_col_idx, &src.col_idx[0], sizeof(int) * src.col_idx.size(), hipMemcpyHostToDevice);

}

template<typename T>
__global__
void d_set_val(T *m, T value, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {

        for (int i = init; i < DIMV; i += stride) {
            m[i] = value;
        }

    }

}

template<typename T>
__global__
void spmv(T *Y, T *pr, T *csc_val, int *csc_non_zero, int *csc_col_idx, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {

            int begin = csc_non_zero[i];
            int end = csc_non_zero[i + 1];

            T acc = 0.0;

            for (int j = begin; j < end; j++) {
                acc += csc_val[j] * pr[csc_col_idx[j]];
            }

            Y[i] = acc;

        }
    }

}


template<typename T>
__global__
void part_spmv(T *Y, T *pr, T *csc_val, int *csc_non_zero, int *csc_col_idx, bool *update_bitmap, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {

            int begin = csc_non_zero[i];
            int end = csc_non_zero[i + 1];
            
            if(update_bitmap[i] == true){

                T acc = 0.0;

                for (int j = begin; j < end; j++) {
                    acc += csc_val[j] * pr[csc_col_idx[j]];
                }

                Y[i] = acc;
            }

        }
    }

}


template<typename T>
__global__
void scale(T *m, T v, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {
            m[i] *= v;
        }
    }
}

template<typename T>
__global__
void shift(T *m, T v, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {
            m[i] += v;
        }
    }
}

template<typename T>
__global__
void compute_error(T *error, T *next, T *prev, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {
            error[i] = abs(next[i] - prev[i]);
        }
    }

}

template<typename T>
__global__
void part_compute_error(T *error, T *next, T *prev, bool *update_bitmap, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV) {
        for (int i = init; i < DIMV; i += stride) {
            if(update_bitmap[i]){
                error[i] = abs(next[i] - prev[i]);
                update_bitmap[i] = error[i] > TAU;
            }
        }
    }

}

__global__
void d_set_dangling_bitmap(bool *dangling_bitmap, int *csc_col_idx, const unsigned DIMV) {

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = init; i < DIMV; i += stride) {
        dangling_bitmap[csc_col_idx[i]] = 0;
    }

}


template<typename T1, typename T2>
T2 dot(size_t n, T1 *x, T2 *y) {
    return thrust::inner_product(thrust::device, x, x + n, y, (T2) 0.0);
}

int amain() {


    /**
     * HOST
     */
    num_type *pr;
    num_type *error;

    /**
     * DEVICE
     */
    num_type *d_pr;
    num_type *d_error;
    num_type *d_spmv_res;
    num_type *d_csc_val;
    int      *d_csc_non_zero;
    int      *d_csc_col_idx;
    bool     *d_dangling_bitmap;
    bool     *d_update_bitmap;

    csc_t csc_matrix = parse_dir("/home/fra/University/HPPS/Approximate-PR/graph_generator/generated_csc/smw");

    const unsigned NON_ZERO = csc_matrix.val.size();
    const unsigned DIM = csc_matrix.non_zero.size() - 1;

    std::cout << "\nFEATURES: " << std::endl;
    std::cout << "\tNumber of non zero elements: " << NON_ZERO << std::endl;
    std::cout << "\tNumber of nodes: " << DIM << std::endl;
    std::cout << "\tSparseness: " << (1 - (((double) NON_ZERO) / (DIM * DIM))) * 100 << "%\n" << std::endl;

    hipHostMalloc(&pr, sizeof(num_type) * DIM);
    hipHostMalloc(&error, sizeof(num_type) * DIM);

    std::cout << "Initializing device memory" << std::endl;

    // Create device memory
    hipMalloc(&d_csc_val, sizeof(num_type) * NON_ZERO);
    hipMalloc(&d_csc_non_zero, sizeof(int) * (DIM + 1));
    hipMalloc(&d_csc_col_idx, sizeof(num_type) * NON_ZERO);
    hipMalloc(&d_pr, sizeof(num_type) * DIM);
    hipMalloc(&d_error, sizeof(num_type) * DIM);
    hipMalloc(&d_spmv_res, sizeof(num_type) * DIM);
    hipMalloc(&d_dangling_bitmap, DIM * sizeof(bool));
    hipMalloc(&d_update_bitmap, DIM * sizeof(bool));

    std::cout << "Parsing csc files" << std::endl;

    to_device_csc(d_csc_val, d_csc_non_zero, d_csc_col_idx, csc_matrix);
    
    std::cout << "Initializing pr, error, dangling bitmap vectors" << std::endl;

    // Initialize error and pr vector
    hipMemset(d_pr, (num_type) 1.0 / DIM, DIM);
    hipMemset(d_error,  (num_type) 1.0, DIM);
    hipMemset(d_dangling_bitmap, true, DIM);
    hipMemset(d_update_bitmap, true, DIM);
    
    d_set_dangling_bitmap << < MAX_B, MAX_T >> > (d_dangling_bitmap, d_csc_col_idx, NON_ZERO);


    // Copy them back to their host vectors
    hipMemcpy(pr, d_pr, DIM * sizeof(num_type), hipMemcpyDeviceToHost);
    hipMemcpy(error, d_error, DIM * sizeof(num_type), hipMemcpyDeviceToHost);

    std::cout << "Beginning pagerank..." << std::endl;

    int iterations = 0;
    bool converged = false;
    while (!converged && iterations < MAX_ITER) {

        // spmv << < MAX_B, MAX_T >> > (d_spmv_res, d_pr, d_csc_val, d_csc_non_zero, d_csc_col_idx, DIM);
        part_spmv <<< MAX_B, MAX_T >>> (d_spmv_res, d_pr, d_csc_val, d_csc_non_zero, d_csc_col_idx, d_update_bitmap, DIM);
        scale << < MAX_B, MAX_T >> > (d_spmv_res, (num_type) ALPHA, DIM);

        // Figure out a way to do the dot product inside GPU
        num_type res_v = dot(DIM, d_dangling_bitmap, d_pr);

        shift << < MAX_B, MAX_T >> > (d_spmv_res, static_cast<num_type> ((1.0 - ALPHA) / DIM + (ALPHA / DIM) * res_v), DIM);

        // compute_error << < MAX_B, MAX_T >> > (d_error, d_spmv_res, d_pr, DIM);
        part_compute_error << < MAX_B, MAX_T >> > (d_error, d_spmv_res, d_pr, d_update_bitmap, DIM);

        hipDeviceSynchronize();

        hipMemcpy(error, d_error, DIM * sizeof(num_type), hipMemcpyDeviceToHost);
        hipMemcpy(d_pr, d_spmv_res, DIM * sizeof(num_type), hipMemcpyDeviceToDevice);

        converged = check_error(error, (num_type) TAU, DIM);

        iterations++;
    }

    hipMemcpy(pr, d_pr, DIM * sizeof(num_type), hipMemcpyDeviceToHost);


    std::cout << "converged after n_iter: " << iterations << std::endl;


    std::map<int, num_type> pr_map;
    std::vector<std::pair<int, num_type>> sorted_pr;
    std::vector<int> sorted_pr_idxs;

    for (int i = 0; i < DIM; ++i) {
        sorted_pr.push_back({i, pr[i]});
        pr_map[i] = pr[i];
        //std::cout << "Index: " << i << " => " << pr_map[i] << std::endl;
    }

    std::sort(sorted_pr.begin(), sorted_pr.end(),
              [](const std::pair<int, num_type> &l, const std::pair<int, num_type> &r) {
                  if(l.second != r.second)return l.second > r.second;
                  else return l.first > r.first;
              });

    // print the vector
    for (auto const &pair: sorted_pr) {
        sorted_pr_idxs.push_back(pair.first);
    }

    std::cout << "Checking results..." << std::endl;

    std::ifstream results;
    results.open("/home/fra/University/HPPS/Approximate-PR/graph_generator/generated_csc/smw/results.txt");

    int i = 0;
    int tmp = 0;
    int errors = 0;

    while (results >> tmp) {
        // std::cout << "reading " << tmp << std::endl;
        if (tmp != sorted_pr_idxs[i]) {
            errors++;
            // std::cout << "ERROR AT INDEX " << i << ": " << tmp << " != " << sorted_pr_idxs[i] << " Value => " << (num_type) pr_map[sorted_pr_idxs[i]] << std::endl;
        }
        i++;
    }

    std::cout << "Percentage of error: " << (((double) errors) / (DIM)) * 100 << "%\n" << std::endl;

    hipFree(&pr);
    hipFree(&error);

    hipFree(&d_pr);
    hipFree(&d_error);
    hipFree(&d_spmv_res);
    hipFree(&d_csc_val);
    hipFree(&d_csc_non_zero);
    hipFree(&d_csc_col_idx);

    return 0;
}
