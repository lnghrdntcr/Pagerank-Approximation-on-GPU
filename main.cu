#include "hip/hip_runtime.h"
//
// Created by Francesco Sgherzi on 15/04/19.
//

#include <iostream>
#include <map>
#include <stdlib.h>
#include <time.h>
#include <vector>
#include <fstream>
#include <set>
#include <math.h>
#include <algorithm>
#include "Parse/Parse.h"

#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/sort.h>

#define TAU 1e-15
#define ALPHA 0.85

#define MAX_B 1024
#define MAX_T 1024

#define MAX_ITER 200

#define num_type double

using namespace std;

template<typename T>
void generate_sparse_matrix(T *matrix, const unsigned int DIMV, const unsigned int min_sparse) {

    // for all rows
    for (int i = 0; i < DIMV; ++i) {

        int num_zeroes = rand() % (DIMV - min_sparse) + min_sparse;
        std::set<int> zero_idxs;

        zero_idxs.insert(i);
        for (int j = 0; j < num_zeroes; ++j) {
            int r_idx = rand() % DIMV;
            zero_idxs.insert(r_idx);
        }

        for (int j = 0; j < DIMV; ++j) {
            if (zero_idxs.find(j) == zero_idxs.end() && (DIMV - zero_idxs.size()) != 0) {
                matrix[i * DIMV + j] = (T) 1.0 / (DIMV - zero_idxs.size());
            }
        }

    }
}

template <typename T>
void fill_spm(T *matrix, const unsigned int DIMV){
    for (int i = 0; i < DIMV; ++i) {
        int count_zero = 0;
        for (int j = 0; j < DIMV; ++j) {
            if(matrix[i * DIMV + j] == 0.0) count_zero++;
        }
        if(count_zero == DIMV) matrix[i * DIMV + i] = 1;
    }
}

template <typename T>
void transpose(T *out, T *in, const unsigned DIMV){

    for (int i = 0; i < DIMV; ++i) {
        for (int j = 0; j < DIMV; ++j) {
            out[i * DIMV + j] = in[j * DIMV + i];
        }
    }

}

template <typename T>
void to_csc(T *csc_val, int *csc_non_zero, int *csc_col_idx, T* src, const unsigned DIMV, const unsigned non_zero){

    unsigned val_idx = 0;

    csc_non_zero[0] = 0;

    for (int i = 0; i < DIMV; ++i) {

        csc_non_zero[i + 1] = csc_non_zero[i];

        for (int j = 0; j < DIMV; ++j) {

            if(src[i * DIMV + j] > 0){
                csc_val[val_idx] = src[i * DIMV + j];
                csc_non_zero[i + 1]++;
                csc_col_idx[val_idx] = j;

                val_idx++;
            }

        }

    }

    cout << "Bella" << endl;

}
template <typename T>
void to_device_csc(T *csc_val, int *csc_non_zero, int *csc_col_idx, const csc_t src){

    hipMemcpy(csc_val, &src.val[0], sizeof(T) * src.val.size(), hipMemcpyHostToDevice);
    hipMemcpy(csc_non_zero, &src.non_zero[0], sizeof(int) * src.non_zero.size(), hipMemcpyHostToDevice);
    hipMemcpy(csc_col_idx, &src.col_idx[0], sizeof(int) * src.col_idx.size(), hipMemcpyHostToDevice);

}

template<typename T>
unsigned int count_non_zero(T *m, const unsigned int DIMV) {
    int sum = 0;

    for (int i = 0; i < DIMV * DIMV; ++i) {
        if (m[i] > 0) sum++;
    }

    return sum;
}

template <typename T>
__global__
void d_set_val( T * m, T value, const unsigned DIMV){

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if(init < DIMV){

        for (int i = init; i < DIMV; i += stride) {
            m[i] = value;
        }

    }

}

template<typename T>
bool check_error(T *e, const T error, const unsigned DIMV) {
    for (int i = 0; i < DIMV; ++i) {
        if(e[i] > error) return false;
    }
    return true;
}

template <typename T>
__global__
void spmv(T *Y, T *pr, T *csc_val, int *csc_non_zero, int *csc_col_idx, const unsigned DIMV){

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if(init < DIMV){
        for(int i = init; i < DIMV; i += stride){

            int begin = csc_non_zero[i];
            int end   = csc_non_zero[i + 1];

            T acc = 0.0;

            for(int j = begin; j < end; j++){
                acc += csc_val[j] * pr[csc_col_idx[j]];
            }

            Y[i] = acc;

        }
    }

}

template <typename T>
__global__
void scale(T *m, T v, const unsigned DIMV){

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if(init < DIMV){
        for (int i = init; i < DIMV; i += stride) {
            m[i] *= v;
        }
    }
}

template <typename T>
__global__
void shift(T *m, T v, const unsigned DIMV){

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if(init < DIMV){
        for (int i = init; i < DIMV; i += stride) {
            m[i] +=v;
        }
    }
}

template <typename T>
__global__
void compute_error(T *error, T *next, T *prev, const unsigned DIMV){

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if(init < DIMV){
        for (int i = init; i < DIMV; i += stride) {
            error[i] = abs(next[i] - prev[i]);
        }
    }

}

__global__
void d_set_dangling_bitmap(bool *dangling_bitmap, int *csc_non_zero, const unsigned DIMV){

    int init = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (init < DIMV){
        for (int i = init; i < DIMV; i += stride) {
            dangling_bitmap[csc_non_zero[i]] = 0;
        }
    }

}

// Tnx parra
template <typename T1, typename T2>
T2 dot(size_t n, T1 *x, T2 *y){
    T2 result = thrust::inner_product(
            thrust::device_pointer_cast(x),
            thrust::device_pointer_cast(x + n),
            thrust::device_pointer_cast(y),
            0.0f);
    return result;
}

int main(){


    /**
     * HOST
     */
    num_type *matrix;
    num_type *matrix_t;
    num_type *pr;
    num_type *spmv_res;
    num_type *error;
    num_type *csc_val;
    int      *csc_non_zero;
    int      *csc_col_idx;


    /**
     * DEVICE
     */
    num_type *d_pr;
    num_type *d_error;
    num_type *d_spmv_res;
    num_type *d_csc_val;
    int      *d_csc_non_zero;
    int      *d_csc_col_idx;
    bool     *d_dangling_bitmap;

    csc_t csc_matrix = parse_dir("/home/fra/University/HPPS/Approximate-PR/graph_generator/generated_csc/test");

    const unsigned NON_ZERO = csc_matrix.val.size();
    const unsigned DIM = csc_matrix.non_zero.size() - 1;

    std::cout << "\nFEATURES: " << std::endl;
    std::cout << "\tNumber of non zero elements: " << NON_ZERO << std::endl;
    std::cout << "\tNumber of nodes: " << DIM << std::endl;
    std::cout << "\tSparseness: " << (1 - (((double) NON_ZERO) / (DIM * DIM))) * 100 << "%\n" << std::endl;

    hipHostMalloc(&matrix, sizeof(num_type) * DIM * DIM);
    hipHostMalloc(&matrix_t, sizeof(num_type) * DIM * DIM);
    hipHostMalloc(&pr, sizeof(num_type) * DIM);
    hipHostMalloc(&spmv_res, sizeof(num_type) * DIM);
    hipHostMalloc(&error, sizeof(num_type) * DIM);


    std::cout << "Initializing device memory" << std::endl;

    // Create device memory
    hipMalloc(&d_csc_val, sizeof(num_type) * NON_ZERO);
    hipMalloc(&d_csc_non_zero, sizeof(int) * (DIM + 1));
    hipMalloc(&d_csc_col_idx, sizeof(num_type) * NON_ZERO);
    hipMalloc(&d_pr, sizeof(num_type) * DIM);
    hipMalloc(&d_error, sizeof(num_type) * DIM);
    hipMalloc(&d_spmv_res, sizeof(num_type) * DIM);
    hipMalloc(&d_dangling_bitmap, DIM * sizeof(bool));

    std::cout << "Parsing csc files" << std::endl;

    to_device_csc(d_csc_val, d_csc_non_zero, d_csc_col_idx, csc_matrix);

    std::cout << "Initializing pr, error, dangling bitmap error" << std::endl;

    // Initialize error and pr vector
    d_set_val<<<MAX_B, MAX_T>>>(d_pr, 1.0 / DIM, DIM);
    d_set_val<<<MAX_B, MAX_T>>>(d_error, 1.0, DIM);
    d_set_val<<<MAX_B, MAX_T>>>(d_dangling_bitmap, true, DIM);
    d_set_dangling_bitmap<<<MAX_B, MAX_T>>>(d_dangling_bitmap, d_csc_non_zero, DIM);

    // Copy them back to their host vectors
    hipMemcpy(pr, d_pr,  DIM * sizeof(num_type), hipMemcpyDeviceToHost);
    hipMemcpy(error, d_error, DIM * sizeof(num_type), hipMemcpyDeviceToHost);


    /**
    * TEST
    */

    std::cout << "Beginning pagerank..." << std::endl;

    int iterations = 0;
    while(!check_error(error, TAU, DIM) && iterations < MAX_ITER){

        // TODO: andare a guardare quali sono i valori ottimali sulla gpu
        spmv<<<MAX_B, MAX_T>>>(d_spmv_res, d_pr, d_csc_val, d_csc_non_zero, d_csc_col_idx, DIM);
        scale<<<MAX_B, MAX_T>>>(d_spmv_res, ALPHA, DIM);

        num_type res_v = dot(DIM, d_dangling_bitmap, d_pr);

        // std::cout << res_v << std::endl;

        shift<<<MAX_B, MAX_T>>>(d_spmv_res, (1.0 - ALPHA) / DIM + (ALPHA / DIM) * /*res_v*/ 0.0, DIM);

        compute_error<<<MAX_B, MAX_T>>>(d_error, d_spmv_res, d_pr, DIM);

        hipDeviceSynchronize();

        hipMemcpy(error, d_error, DIM * sizeof(num_type), hipMemcpyDeviceToHost);
        hipMemcpy(d_pr, d_spmv_res, DIM * sizeof(num_type), hipMemcpyDeviceToDevice);

        iterations++;
    }

    hipMemcpy(pr, d_pr,  DIM * sizeof(num_type), hipMemcpyDeviceToHost);


    std::cout << "converged after n_iter: " << iterations << std::endl;


    std::map <int, num_type> pr_map;
    std::vector<std::pair<int, num_type>> sorted_pr;
    std::vector<int> sorted_pr_idxs;

    for (int i = 0; i < DIM; ++i) {
        sorted_pr.push_back({i, pr[i]});
        pr_map[i] = pr[i];
        //std::cout << "Index: " << i << " => " << pr_map[i] << std::endl;
    }

    std::sort(sorted_pr.begin(), sorted_pr.end(),
              [](const pair<int, num_type>& l, const pair<int, num_type>& r) {
                  return l.second > r.second;
              });

    // print the vector
    for (auto const &pair: sorted_pr) {
        sorted_pr_idxs.push_back(pair.first);
    }

    std::cout << "Checking results..." << std::endl;

    std::ifstream results;
    results.open("/home/fra/University/HPPS/Approximate-PR/graph_generator/generated_csc/test/results.txt");

    int i = 0;
    int tmp = 0;

    while(results >> tmp){
        if(tmp != sorted_pr_idxs[i]){
            std::cout << "ERROR AT INDEX " << i << ": " << tmp << " != " << sorted_pr_idxs[i] << " Value => " << (double) pr_map[sorted_pr_idxs[i]] << std::endl;
        }
        i++;
    }


    hipFree(&matrix);
    hipFree(&matrix_t);
    hipFree(&pr);
    hipFree(&spmv_res);
    hipFree(&error);
    hipFree(&csc_val);
    hipFree(&csc_non_zero);
    hipFree(&csc_col_idx);

    hipFree(&d_pr);
    hipFree(&d_error);
    hipFree(&d_spmv_res);
    hipFree(&d_csc_val);
    hipFree(&d_csc_non_zero);
    hipFree(&d_csc_col_idx);

    return 0;
}